#include "hip/hip_runtime.h"
#include "circulant.hpp"
#include "structured/lib/ExecutiveCore.h"
#include "structured/lib/caffe/util/math_functions.hpp"
#include <vector>
#include <thrust/complex.h>

using namespace structured;
using namespace std;
using namespace caffe;

template <typename Dtype>
__global__ void bat_vmul_knl(const int m, const int n, Dtype* t, const Dtype* r, const Dtype* d) {
  CUDA_KERNEL_LOOP_2D(batch, index, m, n) {
    int off = batch * n;
    (t + off)[index] = (r + off)[index] * d[index];
  }
}

template <typename Dtype>
__global__ void bat_amul_knl(const int m, const int n, Dtype* t, const Dtype* r, const Dtype alpha, const int tn, const int rn) {
  CUDA_KERNEL_LOOP_2D(batch, index, m, n) {
    (t + batch*tn)[index] = alpha * (r + batch*rn)[index];
  }
}

template <typename Dtype>
__global__ void bat_vamul_knl(const int m, const int n, Dtype* t, const Dtype* r, const Dtype* d, const Dtype alpha, const int tn, const int rn) {
  CUDA_KERNEL_LOOP_2D(batch, index, m, n) {
    (t + batch*tn)[index] = alpha * (r + batch*rn)[index] * d[index];
  }
}

template <typename TCore, typename Dtype>
void CirculantProjection<TCore, Dtype>::Compute(
   const Dtype* D_, const int M_, const int K_, const int N_,
   const Dtype* bottom_data,
   Dtype* top_data,
   const Dtype* weight,
   complex<Dtype>* conv_buffer,
   complex<Dtype>* param_buffer,
   Dtype* data_buffer) {

  int Kc = K_ / 2 + 1;
  
  bat_vmul_knl<Dtype><<<CAFFE_GET_BLOCKS_2D(M_, K_), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, K_, data_buffer, bottom_data, D_);
  CUDA_POST_KERNEL_CHECK;
  
  cerr<<"Forward/GPU_FFT"<<endl;
  caffe_gpu_fft<Dtype>(1, K_, weight, param_buffer);
  caffe_gpu_fft<Dtype>(M_, K_, data_buffer, conv_buffer);
  cerr<<"Forward/MUL"<<endl;

  bat_vmul_knl<thrust::complex<Dtype> >
    <<<CAFFE_GET_BLOCKS_2D(M_, Kc), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, Kc,
     reinterpret_cast<thrust::complex<Dtype> *>(conv_buffer),
     reinterpret_cast<thrust::complex<Dtype> *>(conv_buffer),
     reinterpret_cast<thrust::complex<Dtype> *>(param_buffer)
     );
  CUDA_POST_KERNEL_CHECK;
  
  cerr<<"FORWARD/IFFT"<<endl;
  caffe_gpu_ifft<Dtype>(M_, K_, conv_buffer, data_buffer);

  bat_amul_knl<Dtype><<<CAFFE_GET_BLOCKS_2D(M_, K_), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, N_, top_data, data_buffer, (Dtype)1./K_, N_, K_);
  CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
__global__ void bat_cirvmul_knl(const int m, const int n, Dtype* t, const Dtype* r, const Dtype* d) {
  CUDA_KERNEL_LOOP_2D(batch, index, m, n) {
    int off = batch * n;
    (t + off)[(n-index)%n]=(r + off)[index] * d[index];
  }
}

template <typename Dtype>
__global__ void circpy_knl(const int n, Dtype* dist, const Dtype* src) {
  CUDA_KERNEL_LOOP(i, n) {
    dist[(n-i)%n] = src[i];
  }
}

template <typename TCore, typename Dtype>
void CirculantProjection<TCore, Dtype>::GradientOfInput(
   const Dtype* D_, const int M_, const int K_, const int N_,
   const Dtype* top_diff,
   const Dtype* param_data,
   Dtype* grad,
   complex<Dtype>* conv_buffer,
   complex<Dtype>* diff_buffer,
   complex<Dtype>* param_buffer,
   Dtype* data_buffer,
   Dtype* weight_buffer) {

  const int Kc = K_ / 2 + 1;

  circpy_knl<Dtype><<<CAFFE_GET_BLOCKS(K_), CAFFE_CUDA_NUM_THREADS>>>
    (K_,  weight_buffer, param_data);
  CUDA_POST_KERNEL_CHECK;
  caffe_gpu_fft<Dtype>(1, K_, weight_buffer, param_buffer);
  bat_vmul_knl<thrust::complex<Dtype> >
    <<<CAFFE_GET_BLOCKS_2D(M_, Kc), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, Kc,
     reinterpret_cast<thrust::complex<Dtype> *>(conv_buffer),
     reinterpret_cast<thrust::complex<Dtype> *>(diff_buffer),
     reinterpret_cast<thrust::complex<Dtype> *>(param_buffer)
     );
  CUDA_POST_KERNEL_CHECK;
  caffe_gpu_ifft<Dtype>(M_, K_, conv_buffer, data_buffer);
  bat_vamul_knl<Dtype><<<CAFFE_GET_BLOCKS_2D(M_, K_), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, K_, grad, data_buffer, D_, (Dtype)1./K_, K_, K_);
  CUDA_POST_KERNEL_CHECK;

    // Gradient with respect to bottom data

}

template <typename TCore, typename Dtype>
void CirculantProjection<TCore, Dtype>::GradientOfParameter(
   const Dtype* D_, const int M_, const int K_, const int N_,
   const Dtype* top_diff,
   const Dtype* bottom_data,
   Dtype* grad,
   complex<Dtype>* conv_buffer,
   complex<Dtype>* diff_buffer,
   Dtype* data_buffer,
   Dtype* bias_multiplier) {

  const int Kc = K_ / 2 + 1;
  /*
  if (this->param_propagate_down_[0] || propagate_down[0] ){
  
  if (this->param_propagate_down_[0]) {
  */

  cerr<<"Backward/FFT"<<endl;

  bat_cirvmul_knl<Dtype>
      <<<CAFFE_GET_BLOCKS_2D(M_, K_), CAFFE_CUDA_NUM_THREADS_2D>>>
    (M_, K_, data_buffer, bottom_data, D_);
  CUDA_POST_KERNEL_CHECK;
  caffe_gpu_fft<Dtype>(M_, K_, data_buffer, conv_buffer);
  caffe_gpu_mul<complex<Dtype> >(M_ * Kc, conv_buffer, diff_buffer, conv_buffer);
  caffe_gpu_ifft<Dtype>(M_, K_, conv_buffer, data_buffer);
  caffe_gpu_gemv<Dtype>(CblasTrans, M_, K_, (Dtype)1./K_, data_buffer,
			bias_multiplier, (Dtype)0.,
			grad);

}

template class CirculantProjection<GpuCore, float>;
template class CirculantProjection<GpuCore, double>;
